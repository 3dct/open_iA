#include "hip/hip_runtime.h"
//#include <cutil.h>

extern "C" void init_cuda(int argc, char **argv);
extern "C" void node_texture_bind(const void *data, size_t size);
extern "C" void node_texture_unbind();
extern "C" void tri_texture_bind(const void *data, size_t size);
extern "C" void tri_texture_unbind();
extern "C" void id_texture_bind(const void *data, size_t size);
extern "C" void id_texture_unbind();
extern "C" void normal_texture_bind(const void *data, size_t size);
extern "C" void normal_texture_unbind();
extern "C" void cuda_raycast(void *a_aabb, const void * a_o, const void * a_c, const void * a_dx, const void * a_dy, int w, int h, float* out_res, float * out_dip_res, void * a_cut_aabbs, unsigned int a_cut_aabbs_count);
extern "C" void cuda_raycast_batch(void *a_aabb, void * a_o, void * a_c, void * a_dx, void * a_dy, int w, int h, float* out_res, float * out_dip_res, unsigned int batchSize, void * a_cut_aabbs, unsigned int cut_aabbs_count);
extern "C" void cuda_init(int w, int h, unsigned int batchSize);
extern "C" void cuda_terminate();

#define THREAD_W 1
#define THREAD_H 32

#define MAX_BATCH_SIZE 500
#define MAX_CUT_AAB_COUNT 10
#define MAX_ISEC_COUNT 24

void *device_out_data = 0;
void *device_out_dip = 0;
unsigned int out_size;
/* Texture for kd-tree nodes */
texture<unsigned int, 1, hipReadModeElementType> node_tex;

/* Texture for triangles */
texture<float4, 1, hipReadModeElementType> tri_tex;

/* Texture for ids */
texture<unsigned int, 1, hipReadModeElementType> id_tex;

/* Texture to output resultats */
//texture<float, 1, hipReadModeElementType> out_tex;
/***************************************************************************//**
* Initialize the node texture
/******************************************************************************/
void node_texture_bind(const void *data, size_t size)
{
	node_tex.addressMode[0] = hipAddressModeWrap;
	node_tex.addressMode[1] = hipAddressModeWrap;
	node_tex.filterMode = hipFilterModePoint;
	node_tex.normalized = false;
	hipBindTexture(0, node_tex, data, size);
}
void node_texture_unbind()
{
	hipUnbindTexture(node_tex);
}
/***************************************************************************//**
* Initialize the triangle texture
/******************************************************************************/
void tri_texture_bind(const void *data, size_t size)
{
	tri_tex.addressMode[0] = hipAddressModeWrap;
	tri_tex.addressMode[1] = hipAddressModeWrap;
	tri_tex.filterMode = hipFilterModePoint;
	tri_tex.normalized = false;
	hipBindTexture(0, tri_tex, data, size);
}
void tri_texture_unbind()
{
	hipUnbindTexture(tri_tex);
}
/***************************************************************************//**
* Initialize the id texture
/******************************************************************************/
void id_texture_bind(const void *data, size_t size)
{
	id_tex.addressMode[0] = hipAddressModeWrap;
	id_tex.addressMode[1] = hipAddressModeWrap;
	id_tex.filterMode = hipFilterModePoint;
	id_tex.normalized = false;
	hipBindTexture(0, id_tex, data, size);
}
void id_texture_unbind()
{
	hipUnbindTexture(id_tex);
}

/**	\struct iAVec3.
\brief Class representing 3 dimensional vector. CUDA version.
*/
struct iAVec3
{
public:
	float x,y,z;
	__device__ iAVec3(){}
	__device__ iAVec3(float px, float py, float pz) :x(px), y(py), z(pz)	{}
	//__device__ iAVec3(const iAVec3& v):x(v.x), y(v.y), z(v.z)	{}
	__device__ iAVec3& operator= (const iAVec3& v) { x=v.x;	y=v.y; z=v.z;	return *this; }
	__device__ iAVec3 operator+ () const { return *this; }
	__device__ iAVec3 operator- () const { return iAVec3(-x,-y,-z); }
	__device__ iAVec3& operator+= (const iAVec3& v)	{ x+=v.x; y+=v.y; z+=v.z;	return *this; }
	__device__ iAVec3& operator-= (const iAVec3& v)	{ x-=v.x; y-=v.y; z-=v.z;	return *this; }
	__device__ iAVec3& operator*= (const iAVec3& v) { x*=v.x; y*=v.y; z*=v.z;	return *this; }
	__device__ iAVec3& operator*= (float f)       { x*=f; y*=f; z*=f;			return *this; }
	__device__ iAVec3& operator/= (const iAVec3& v)	{ x/=v.x; y/=v.y; z/=v.z;	return *this; }
	__device__ const float& operator[] (int index) const { return *(index+&x); }
	__device__ float& operator[] (int index) { return *(index+&x); }
	__device__ int operator== (const iAVec3& v) const	{ return x==v.x&&y==v.y&&z==v.z; }
	__device__ int operator!= (const iAVec3& v) const	{ return x!=v.x||y!=v.y||z!=v.z; }
	__device__ int operator<  (const iAVec3& v) const	{ return ( x < v.x ) || ((x == v.x) && (y < v.y)); }
	__device__ int	    operator>  (const iAVec3& v) const { return ( x > v.x ) || ((x == v.x) && (y > v.y)); }
	__device__ float	length () const	{ return (float) sqrt(x*x + y*y + z*z);	}
	/*
	friend __device__ iAVec3 operator + (const iAVec3&,const iAVec3&);
		friend __device__ iAVec3 operator - (const iAVec3&,const iAVec3&);
		friend __device__ iAVec3 operator * (const iAVec3&,const iAVec3&);
		friend __device__ iAVec3 operator * (float          ,const iAVec3&);
		friend __device__ iAVec3 operator * (const iAVec3&,float);
		friend __device__ iAVec3 operator / (const iAVec3&,float);
		friend __device__ iAVec3 operator / (const iAVec3&,const iAVec3&);
		friend __device__ float    operator & (const iAVec3&,const iAVec3&);
		friend __device__ iAVec3 operator ^ (const iAVec3&,const iAVec3&);*/
	
};
__device__  iAVec3 operator + (const iAVec3& u,const iAVec3& v) {	return iAVec3(u.x + v.x, u.y + v.y, u.z + v.z); }
__device__  iAVec3 operator - (const iAVec3& u,const iAVec3& v) {	return iAVec3(u.x - v.x, u.y - v.y, u.z - v.z); }
__device__  iAVec3 operator * (const iAVec3& u,const iAVec3& v) { return iAVec3(u.x * v.x, u.y * v.y, u.z * v.z); }
__device__  iAVec3 operator * (const iAVec3& v,float a) { return iAVec3(v.x*a, v.y*a, v.z*a); }
__device__  iAVec3 operator * (float a, const iAVec3& v) { return iAVec3(v.x*a, v.y*a, v.z*a); }
__device__  iAVec3 operator / (const iAVec3& v,float a) { return iAVec3(v.x/a, v.y/a, v.z/a); }
__device__  iAVec3 operator / (const iAVec3& u,const iAVec3& v) {	return iAVec3(u.x / v.x, u.y / v.y, u.z / v.z); }
//dot
__device__  float    operator & (const iAVec3& u,const iAVec3& v) {	return u.x*v.x + u.y*v.y + u.z*v.z; }
//cross
__device__  iAVec3 operator ^ (const iAVec3& u,const iAVec3& v) {	return iAVec3(u.y*v.z-u.z*v.y, u.z*v.x-u.x*v.z, u.x*v.y-u.y*v.x); }
__device__ void normalize_vec3(iAVec3& u)
{
	float len = u.length();
	if(len>0)
		u = u / u.length();
}
/**	\struct ct_state.
	\brief Structure describe current CT state.
*/
struct ct_state{
	iAVec3 o;///< rays origin
	iAVec3 c;///< corner of plate
	iAVec3 dx;///< dx of plane in 3d
	iAVec3 dy;///< dy of plane in 3d
};

/***************************************************************************//**
* The axis-aligned bounding box
/******************************************************************************/
struct aabb {
	float x1,x2,y1,y2,z1,z2;
	int isInside(iAVec3& v) const
	{
		if(v.x<=x2)
			if(v.x>=x1) 
				if(v.y<=y2)
					if(v.y>=y1) 
						if(v.z<=z2)
							if(v.z>=z1)
								return 1;
		return 0;
	}
};

//constants  *******************************************************************/
__constant__ iAVec3 o;
__constant__ iAVec3 c;
__constant__ iAVec3 dx;
__constant__ iAVec3 dy;
//for batching MAX_BATCH_SIZE - is max batch size
__constant__ iAVec3 os[MAX_BATCH_SIZE];
__constant__ iAVec3 cs[MAX_BATCH_SIZE];
__constant__ iAVec3 dxs[MAX_BATCH_SIZE];
__constant__ iAVec3 dys[MAX_BATCH_SIZE];
//__constant__ unsigned int tri_count;
//__constant__ ct_state cur_ctstate;
/* The bounding box of the scene */
__constant__ aabb cu_aabb;
__constant__ unsigned int cut_aabbs_count;
__constant__ aabb cut_aabbs[MAX_CUT_AAB_COUNT];


/**	\struct BSPNode.
\brief Class representing a BSP-tree node.

BSP-tree node.	
*/
class BSPNode
{
#define internal1 internal_data.x
#define internal2 internal_data.y
#define masked_vars internal_data.z
public:
	//
	//char masked_vars; ///< Is this node a leaf-node first bit -- is leaf, has left, has right, else -- axis index
	uint3 internal_data;
	//internal1, internal2,masked_vars;///< shared data, depends if node is leaf or not
	//
	__device__ bool isLeaf() const {return masked_vars&0x00000080;}
	__device__ int axisInd() const { return masked_vars&0x00000003;}
	__device__ void setLeaf(bool a_isLeaf){ masked_vars&=0x0000007f; if(a_isLeaf)masked_vars|=0x00000080; }
	__device__ void setAxisInd(int index){
		switch(index){
			case 0:	 masked_vars|=(char)0x00000000; break;
			case 1:	 masked_vars|=(char)0x00000001; break;
			default: masked_vars|=(char)0x00000002; break;
		}
	}
	__device__ bool has_left() const {return masked_vars&0x00000040;}
	__device__ bool has_right() const {return masked_vars&0x00000020;}
	__device__ void set_has_left (bool has){ masked_vars&=0x000000bf; if(has) masked_vars|=0x00000040;	}
	__device__ void set_has_right(bool has){ masked_vars&=0x000000df; if(has) masked_vars|=0x00000020;	}
	__device__ unsigned int tri_start() {return internal1;}
	__device__ unsigned int tri_count() {return internal2;}
	__device__ unsigned int offset()    {return internal1;}
	__device__ float & splitCoord()      {return *((float*)&internal2);}
	__device__ void set_tri_start(unsigned int val) { internal1=val; }
	__device__ void set_tri_count(unsigned int val) { internal2=val; }
	__device__ void set_offset(unsigned int val)    { internal1=val; }
	__device__ void set_splitCoord(float val)       { internal2=*((unsigned int*)&val); }
	//__device__ BSPNode *get_left(std::vector<BSPNode*> &nodes)  { return nodes[offset()]; }
	//__device__ BSPNode *get_right(std::vector<BSPNode*> &nodes) { return nodes[offset()+1]; }
	//__device__ BSPNode *get_left(const std::vector<BSPNode*> &nodes)  { return nodes[offset()]; }
	//__device__ BSPNode *get_right(const std::vector<BSPNode*> &nodes) { return nodes[offset()+1]; }
};
/**	\struct traverse_stack.
	\brief Used for tree traversal.

	CUDA version.	
*/
struct trace_t {
	//__device__ trace_t(unsigned int aa, float a, float b):node(aa),tmin(a),tmax(b){}
	unsigned int node;
	float tmin;
	float tmax;
};
struct traverse_stack
{
	int index;
	trace_t t[30];

	//__device__ traverse_stack(int a_size){ size = a_size; index=0; }
	__device__ traverse_stack(): index(0) {}
	__device__ inline void push(unsigned int node,	float tmin,	float tmax)
	{ 
		t[index].node = node; t[index].tmin = tmin; t[index].tmax = tmax; 
		index++;
	}
	__device__ inline trace_t& get() { return t[index-1]; }
	__device__ inline trace_t& pop() { index--; return t[index]; }
	__device__ inline int numElements() { return index; }
};
/**	\struct wald_tri.
\brief Structure representing a triangle data needed for intersection test.

Wald triangle structure.	
*/

struct wald_tri
{
	float4 intern0, intern1, intern2, intern3;

	__device__ iAVec3 m_N(){ return iAVec3(intern0.x, intern0.y, intern0.z);}
	__device__ iAVec3 m_A(){ return iAVec3(intern0.w, intern1.x, intern1.y);}					
	__device__ float nu(){ return intern1.z;}
	__device__ float nv(){ return intern1.w;}
	__device__ float nd(){ return intern2.x;}						
	__device__ unsigned int k(){ return __float_as_int(intern2.y);}										
	__device__ float bnu(){ return intern2.z;}
	__device__ float bnv(){ return intern2.w;}						
	__device__ float cnu(){ return intern3.x;}
	__device__ float cnv(){ return intern3.y;}					
};

/**	\class Ray.
\brief Class representing ray in 3D.

CUDA version.	
*/
class Ray
{
public:
	__device__ Ray() : o( iAVec3( 0.f, 0.f, 0.f ) ), dir( iAVec3( 0, 0, 0 ) ) {}
	__device__ Ray( iAVec3& a_Origin, iAVec3& a_Dir ) {o=a_Origin; dir = a_Dir;}

	iAVec3 o;	///< ray origin's position
	iAVec3 dir;	///< ray direction vector
};
/**	\struct intersection.
\brief Structure representing intersection data.

Contains data about primitive. CUDA version	
*/
struct intersection
{
	unsigned int prim_ind;
	float dist;
};

/***************************************************************************//**
 * Initialize CUDA
/******************************************************************************/
void init_cuda(int argc, char **argv)
{
	//CUT_DEVICE_INIT(argc, argv);
}
/**
* Ray-AABB intersection routine. CUDA version.
* @param ray ray class.
* @param box axis aligned bounding box structure.
* @return 
1 - if ray intersects AABB
0 - otherwise
*/
static __device__ inline unsigned int Intersect(const Ray &ray, const aabb& box, float &tmin, float&tmax)
{
	float l1 = __fdividef(box.x1 - ray.o.x, ray.dir.x);
	float l2 = __fdividef(box.x2 - ray.o.x, ray.dir.x);
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);
	l1 = __fdividef(box.y1 - ray.o.y, ray.dir.y);
	l2 = __fdividef(box.y2 - ray.o.y, ray.dir.y);
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);
	l1 = __fdividef(box.z1 - ray.o.z, ray.dir.z);
	l2 = __fdividef(box.z2 - ray.o.z, ray.dir.z);
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);
	return ((tmax >= tmin) & (tmax >= 0.f));
/*
	float txmin, txmax, tymin, tymax;
	float ddx = 1.0f/ray.dir.x;
	float ddy = 1.0f/ray.dir.y;
	if(ddx>=0)
	{
		txmin = (box.x1 - ray.o.x) * ddx;
		txmax = (box.x2 - ray.o.x) * ddx;
	}
	else
	{
		txmin = (box.x2 - ray.o.x) * ddx;
		txmax = (box.x1 - ray.o.x) * ddx;
	}
	if(ddy>=0)
	{
		tymin = (box.y1 - ray.o.y) * ddy;
		tymax = (box.y2 - ray.o.y) * ddy;
	}
	else
	{
		tymin = (box.y2 - ray.o.y) * ddy;
		tymax = (box.y1 - ray.o.y) * ddy;
	}
	if( (txmin>tymax) || (tymin>txmax) ) return 0;
	if( tymin>txmin ) txmin=tymin;
	if( tymax<txmax ) txmax=tymax;

	float tzmin, tzmax;
	float ddz = 1.0f/ray.dir.z;
	if(ddz>=0)
	{
		tzmin = (box.z1 - ray.o.z) * ddz;
		tzmax = (box.z2 - ray.o.z) * ddz;
	}
	else
	{
		tzmin = (box.z2 - ray.o.z) * ddz;
		tzmax = (box.z1 - ray.o.z) * ddz;
	}
	if( (txmin>tzmax) || (tzmin>txmax) ) return 0;
	if( tzmin>txmin ) txmin=tzmin;
	if( tzmax<txmax ) txmax=tzmax;
	tmin=txmin;
	tmax=txmax;
	return 1;*/

}
/**
* Ray-AABB intersection routine.
* checks which subnodes' AABBs are intersected by ray. CUDA version.
* @param ray ray class.
* @param tmin parent AABB min t.
* @param tmin parent AABB max t.
* @param split split plane's coordinates
* @param splitIndex index of splitting axis
* @param t [out] split plane's t
* @return 
0 - left node intersected
1 - both nodes intersected
2 - right node intersected
*/
__device__ inline int GetIntersectionState(const Ray &ray, float &tmin, float &tmax, float &split, int splitIndex, float &t)
{
	float rd = *(&ray.dir.x+splitIndex);
	if(!rd)
		rd=0.00000001f;
	t = (split - *(&ray.o.x+splitIndex)) / rd;
	const unsigned int sign = (rd >= 0.0f);
	if(t<tmin) return (int)(sign^0);
	if(t>tmax) return (int)(sign^1);
	return 2;
}
/***************************************************************************//**
* Ray vs Wald Tri intersection routine
/******************************************************************************/
// Intersection method return values
#define HIT		 1		// Ray hit primitive
#define MISS	 0		// Ray missed primitive
#define INPRIM	-1		// Ray started inside primitive
#define ku modulo[wt.k() + 1]
#define kv modulo[wt.k() + 2]

__constant__ unsigned int modulo[] = { 0, 1, 2, 0, 1 };
__device__ inline int Intersect(const Ray& a_Ray, wald_tri& wt, float& a_Dist, float& a_Dip )
{
	const float lnd = __fdividef(1.0f, (a_Ray.dir[wt.k()] + wt.nu() * a_Ray.dir[ku] + wt.nv() * a_Ray.dir[kv]));
	const float t = (wt.nd() - a_Ray.o[wt.k()] - wt.nu() * a_Ray.o[ku] - wt.nv() * a_Ray.o[kv]) * lnd;
	if (!(a_Dist > t && t > 0)) return MISS;
	const float hu = a_Ray.o[ku] + t*a_Ray.dir[ku] - wt.m_A()[ku];
	const float hv = a_Ray.o[kv] + t*a_Ray.dir[kv] - wt.m_A()[kv];
	const float beta = hv * wt.bnu() + hu * wt.bnv();//=wt.m_U=//��� ��������
	if (beta < 0) return MISS;
	const float gamma = hu * wt.cnu() + hv * wt.cnv();//=m_WaldTri.m_V=
	if (gamma < 0) return MISS;
	if ((beta + gamma) > 1) return MISS;
	a_Dist = t;
	a_Dip=a_Ray.dir&wt.m_N();
	return ( a_Dip > 0 )? INPRIM : HIT;
}
__device__ inline int Intersect_v2(const Ray& a_Ray, wald_tri& wt, float& a_Dist )
{
	const float lnd = 1.0f / (a_Ray.dir[wt.k()] + wt.nu() * a_Ray.dir[ku] + wt.nv() * a_Ray.dir[kv]);
	const float t = (wt.nd() - a_Ray.o[wt.k()] - wt.nu() * a_Ray.o[ku] - wt.nv() * a_Ray.o[kv]) * lnd;
	if (!(a_Dist > t && t > 0)) return MISS;
	float hu = a_Ray.o[ku] + t*a_Ray.dir[ku] - wt.m_A()[ku];
	float hv = a_Ray.o[kv] + t*a_Ray.dir[kv] - wt.m_A()[kv];
	float beta = hv * wt.bnu() + hu * wt.bnv();//=wt.m_U=//��� ��������
	if (beta < 0) return MISS;
	float gamma = hu * wt.cnu() + hv * wt.cnv();//=m_WaldTri.m_V=
	if (gamma < 0) return MISS;
	if ((beta + gamma) > 1) return MISS;
	a_Dist = t;
	return ((a_Ray.dir&wt.m_N() ) > 0)? INPRIM : HIT;
}

__device__ inline  void ReadTri(wald_tri &tri, const unsigned int & index)
{
	tri.intern0 = tex1Dfetch(tri_tex, 4 * index);
	tri.intern1 = tex1Dfetch(tri_tex, 4 * index+1);
	tri.intern2 = tex1Dfetch(tri_tex, 4 * index+2);
	tri.intern3 = tex1Dfetch(tri_tex, 4 * index+3);
}
/***************************************************************************//**
* Trace the ray inside the tree
/******************************************************************************/
static __device__ inline int trace_tree(const Ray &ray, float &res, float &res_dip)
{
	//__shared__ wald_tri cur_tri[THREAD_W][THREAD_H];
	//unsigned int isec_count=0;
	res = 0;//100000.f;
	res_dip=0;
	float cur_tmin=0;
	float cur_tmax=100000.f;
	if(!Intersect(ray, cu_aabb, cur_tmin, cur_tmax)) return 0;
	//check for cut aabbs
	if(cut_aabbs_count!=0)
	{
		bool intersects = false;
		for (unsigned int i=0; i<cut_aabbs_count; i++)
		{
			float a=0.f, b=100000.f;
			if(Intersect(ray, cut_aabbs[i], a, b)) 
			{
				intersects = true;
				break;
			}
		}
		if(!intersects)
			return 0;
	}
	intersection intersections[MAX_ISEC_COUNT];//intersection intersections[20];
	intersections[0].dist = 0;//intersections[0].dist = res;
	unsigned int isec_count=0;
	traverse_stack tr_stack;
	//const unsigned int resint = Intersect(ray, cu_aabb, cur_tmin, cur_tmax);
	unsigned int cur_node_id=0;
	tr_stack.push(cur_node_id, cur_tmin, cur_tmax);
	BSPNode  cur_node;
	unsigned int sign = 0;
	//unsigned int node_isecs_start=0;
	while (tr_stack.numElements() > 0)
	{
		cur_node_id = tr_stack.get().node;
		cur_tmin = tr_stack.get().tmin;
		cur_tmax = tr_stack.get().tmax;
		tr_stack.pop();
		cur_node.internal1 = tex1Dfetch(node_tex, 3 * cur_node_id);
		cur_node.internal2 = tex1Dfetch(node_tex, 3 * cur_node_id+1);
		cur_node.masked_vars = tex1Dfetch(node_tex, 3 * cur_node_id+2);

		//tmin=cur_tmin; tmax=cur_tmax;
		if(cur_node.isLeaf())
		{
			wald_tri cur_tri;
			//node_isecs_start = isec_count;
			for (unsigned int i=0; i<cur_node.tri_count(); i++)
			{
				float a_Dist = 1000000.0f, a_Dip;
				//ReadTri(cur_tri, cur_node.tri_start()+i);
				const unsigned int indx = tex1Dfetch(id_tex, cur_node.tri_start()+i);
				ReadTri(cur_tri, indx);
				if (Intersect( ray, cur_tri, a_Dist, a_Dip )) 
				{
					unsigned char again=0;
					for (unsigned int i=0; i<isec_count; i++)
					{
						if(intersections[i].prim_ind==indx)
							again=1;
							
					}
					if(again)
						continue;
					//iAVec3 isec = ray.o+ray.dir*a_Dist;			
					intersections[isec_count].prim_ind = indx;
					intersections[isec_count].dist=a_Dist;//checked
					isec_count++;
					res_dip+=fabsf(a_Dip);
					/*intersections[isec_count].prim_ind = cur_node.tri_start()+i;
					intersections[isec_count].dist =a_Dist;//checked
					if(intersections[isec_count].dist<intersections[0].dist)
					{
						intersections[isec_count].prim_ind = intersections[0].prim_ind;
						intersections[isec_count].dist =intersections[0].dist;
						intersections[0].prim_ind = cur_node.tri_start()+i;
						intersections[0].dist = a_Dist;//checked
					}
					isec_count++;*/
				}
			}
		}
		else 
		{
			float t;
			const int resisec = GetIntersectionState(ray, cur_tmin, cur_tmax, cur_node.splitCoord(), cur_node.axisInd(),t);
			switch(resisec)
			{
			case 0://left only
				if(cur_node.has_left())
				{
					tr_stack.push(cur_node.offset(),cur_tmin,cur_tmax);
				}
				break;
			case 1://right only
				if(cur_node.has_right())
				{
					tr_stack.push(cur_node.offset()+1,cur_tmin,cur_tmax);
				}
				break;
			case 2://both
				sign = ray.dir[cur_node.axisInd()]>=0.0f;
				if(sign)
				{
					if(cur_node.has_left())
						tr_stack.push(cur_node.offset(),cur_tmin,t);
					if(cur_node.has_right())
						tr_stack.push(cur_node.offset()+1,t,cur_tmax);
				}
				else
				{
					if(cur_node.has_right())
						tr_stack.push(cur_node.offset()+1,cur_tmin,t);
					if(cur_node.has_left())
						tr_stack.push(cur_node.offset(),t,cur_tmax);	
				}
				/*if(cur_node.has_right())
				{
					if(sign)
						tr_stack.push(cur_node.offset()+1,t,cur_tmax);
					else
						tr_stack.push(cur_node.offset()+1,cur_tmin,t);
				}
				if(cur_node.has_left())
				{
					if(sign)
						tr_stack.push(cur_node.offset(),cur_tmin,t);
					else
						tr_stack.push(cur_node.offset(),t,cur_tmax);					
				}*/
				break;
			}
		}
	}

	//sort intersections in one node
	for(unsigned int i = 0; i < isec_count; i++)
	{
		float tmp;
		unsigned char swapped=0;
		for(unsigned int j = 0; j < isec_count-1-i; j++)
		{
			if( intersections[j].dist > intersections[j+1].dist )
			{
				tmp = intersections[j].dist;
				intersections[j].dist = intersections[j+1].dist;
				intersections[j+1].dist = tmp;
				if(swapped==0) swapped=1;
			}
		}
		if(swapped == 0) break;
	}
	unsigned int n=0;
	if(isec_count % 2 == 0) //TODO: temporary workaround
	for (unsigned int i=0; i<isec_count; i++)
	{
		if((i+n)%2==1)
		{
			//TODO: no triangles repeated?
			if(intersections[i].prim_ind == intersections[i-1].prim_ind)//if intersections belong to the same triangle
				n++;
			else
				res+=intersections[i].dist - intersections[i-1].dist;
		}
	}
	if(isec_count) res_dip /= isec_count;
	//res=intersections.dist;
	//res=intersections[0].dist;
	return 1;
}
/***************************************************************************//**
* Brutforce raycasting
/******************************************************************************/
/*static __device__ inline int brutforce(const Ray &ray, float &res)
{
	intersection isec;
	res = 100000.f;
	isec.dist = res;
	float cur_tmin=0;
	float cur_tmax=100000.f;
	if(!Intersect(ray, cu_aabb, cur_tmin, cur_tmax)) return 0;
	wald_tri cur_tri;
	for (unsigned int i=0; i<tri_count; i++)
	{
		float a_Dist = 1000000.0f;
		ReadTri(cur_tri, i);
		if (Intersect( ray, cur_tri, a_Dist )) 
		{
			//iAVec3 isec = ray.GetOrigin()+ray.GetDirection()*a_Dist;			
			if(a_Dist<isec.dist)
			{
				isec.prim_ind = i;
				isec.dist =a_Dist;//checked
			}
		}
	}
	res=isec.dist;
	return 1;
}*/
/***************************************************************************//**
* Perform the ray casting
/******************************************************************************/
__global__ void raycast(int w, int h, void* a_out_data, void* out_dip)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int bw = blockDim.x;
	const int bh = blockDim.y;
	const int x = blockIdx.x*bw + tx;
	const int y = blockIdx.y*bh + ty;
	
	iAVec3 dir = (c + x*dx + y*dy) - o;
	normalize_vec3( dir );
	Ray r;
	r.o=o;
	r.dir=dir;//( o, dir );
	float res, res_dip;
	trace_tree(r, res, res_dip);
	//hipMemcpy(out_data+(y * w + x)*sizeof(float), &res, sizeof(float), hipMemcpyDeviceToDevice);
	const int xinv=w-1-x;
	((float*)a_out_data)[y * w + xinv]=res;
	((float*)out_dip)[y * w + xinv]=res_dip;
}
/***************************************************************************//**
* Raycast with CUDA
/******************************************************************************/
void cuda_raycast(void *a_aabb, const void * a_o, const void * a_c, const void * a_dx, const void * a_dy, int w, int h, float* out_res, float * out_dip_res, void * a_cut_aabbs, unsigned int a_cut_aabbs_count)
{
	hipError_t res = hipMalloc(&device_out_data, out_size);
	res = hipMalloc(&device_out_dip, out_size);
	dim3 block, grid;
	block = dim3(THREAD_W, THREAD_H, 1);
	grid = dim3(w / block.x, h / block.y, 1);
	hipMemcpyToSymbol(HIP_SYMBOL(o), a_o, sizeof(iAVec3));
	hipMemcpyToSymbol(HIP_SYMBOL(c), a_c, sizeof(iAVec3));
	hipMemcpyToSymbol(HIP_SYMBOL(dx), a_dx, sizeof(iAVec3));
	hipMemcpyToSymbol(HIP_SYMBOL(dy), a_dy, sizeof(iAVec3));
	hipMemcpyToSymbol(HIP_SYMBOL(cu_aabb), a_aabb, sizeof(aabb));
	hipMemcpyToSymbol(HIP_SYMBOL(cut_aabbs_count), &a_cut_aabbs_count, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(cut_aabbs), a_cut_aabbs, sizeof(aabb)*MAX_CUT_AAB_COUNT);//TODO: krivo max size brat'
	raycast<<<grid, block>>>(w, h, device_out_data, device_out_dip);	
	res = hipMemcpy(out_res, device_out_data, out_size, hipMemcpyDeviceToHost);
	res = hipMemcpy(out_dip_res, device_out_dip, out_size, hipMemcpyDeviceToHost);
	res = hipFree(device_out_data);
	device_out_data=0;
	res = hipFree(device_out_dip);
	device_out_dip=0;
}

/***************************************************************************//**
* Perform the batch ray casting
/******************************************************************************/
__global__ void raycast_batch(int w, int h, unsigned int batchSize, void* a_out_data, void* out_dip)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int bw = blockDim.x;
	const int bh = blockDim.y;
	const int x1 = blockIdx.x*bw + tx;
	const int batch = x1 / w;
	const int offset = w*h*batch;
	const int x = x1 % w;
	const int y = blockIdx.y*bh + ty;

	iAVec3 dir = (cs[batch] + x*dxs[batch] + y*dys[batch]) - os[batch];
	normalize_vec3( dir );
	Ray r;
	r.o=os[batch];
	r.dir=dir;//( o, dir );
	float res, res_dip;
	trace_tree(r, res, res_dip);
	//hipMemcpy(out_data+(y * w + x)*sizeof(float), &res, sizeof(float), hipMemcpyDeviceToDevice);
	const int xinv = w - x - 1; //inv for inteverion)
	((float*)a_out_data)[offset + y * w + xinv] = res;
	((float*)out_dip)[offset + y * w + xinv] = res_dip;
}
/***************************************************************************//**
* Raycast batch with CUDA
/******************************************************************************/
//TODO: handling restrictions, maybe should return error code
void cuda_raycast_batch(void *a_aabb, void * a_o, void * a_c, void * a_dx, void * a_dy, int w, int h, float* out_res, float * out_dip_res, unsigned int batchSize, void * a_cut_aabbs, unsigned int a_cut_aabbs_count)
{
	hipError_t res = hipMalloc(&device_out_data, out_size);
	res = hipMalloc(&device_out_dip, out_size);
	dim3 block, grid;
	block = dim3(THREAD_W, THREAD_H, 1);
	grid = dim3(w*batchSize / block.x, h / block.y, 1);
	hipMemcpyToSymbol(HIP_SYMBOL(os), a_o, sizeof(iAVec3)*batchSize);
	hipMemcpyToSymbol(HIP_SYMBOL(cs), a_c, sizeof(iAVec3)*batchSize);
	hipMemcpyToSymbol(HIP_SYMBOL(dxs), a_dx, sizeof(iAVec3)*batchSize);
	hipMemcpyToSymbol(HIP_SYMBOL(dys), a_dy, sizeof(iAVec3)*batchSize);
	hipMemcpyToSymbol(HIP_SYMBOL(cu_aabb), a_aabb, sizeof(aabb));
	hipMemcpyToSymbol(HIP_SYMBOL(cut_aabbs_count), &a_cut_aabbs_count, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(cut_aabbs), a_cut_aabbs, sizeof(aabb)*MAX_CUT_AAB_COUNT);//TODO: krivo max size brat'
	//hipMemcpyToSymbol(HIP_SYMBOL(tri_count), a_tri_count, sizeof(unsigned int));
	raycast_batch<<<grid, block>>>(w, h, batchSize, device_out_data, device_out_dip);	
	res = hipMemcpy(out_res, device_out_data, out_size, hipMemcpyDeviceToHost);
	res = hipMemcpy(out_dip_res, device_out_dip, out_size, hipMemcpyDeviceToHost);
	res = hipFree(device_out_data);
	device_out_data=0;
	res = hipFree(device_out_dip);
	device_out_dip=0;
	//raycast<<<grid, block, 100*(4*sizeof(iAVec3)+sizeof(cu_aabb))>>>(w, h);
}
void cuda_init(int w, int h, unsigned int batchSize)
{
	out_size = w*h*sizeof(float)*batchSize;
	//hipMalloc(&device_out_data, out_size);
	//hipMalloc(&device_out_dip, out_size);
}
void cuda_terminate()
{
	if(device_out_data)
	{
		hipFree(device_out_data);
		device_out_data=0;
	}
	if(device_out_dip)
	{
		hipFree(device_out_dip);
		device_out_dip=0;
	}
}